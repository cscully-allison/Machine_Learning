#include "hip/hip_runtime.h"
#include "ecos.h"
#include "data.h"

__global__ void ECOSCall(idxint n,idxint m,idxint p,idxint l,idxint ncones,idxint* q, idxint e, pfloat* Gpr ,idxint* Gjc, idxint* Gir, pfloat* Apr, idxint* Ajc, idxint* Air, pfloat* c, pfloat* h, pfloat* b){
    idxint exitflag = ECOS_FATAL;
    pwork* mywork;

    /* set up data */
    mywork = ECOS_setup(n, m, p, l, ncones, q, 0, Gpr, Gjc, Gir, Apr, Ajc, Air, c, h, b);

    if( mywork != NULL ){

        /* solve */
        exitflag = ECOS_solve(mywork);

        /* clean up memory */
        ECOS_cleanup(mywork, 0);

    }

    /* test version number
    ECOS_ver(ver);
    printf("This test has been run on ECOS version %s\n", ver);
     */

    /* explicitly truncate exit code */
    //return (int)exitflag;

}


int main(){
    /*char ver[7];*/




    return 0;
}
