/* Produced by CVXGEN, 2018-04-03 18:09:48 -0400.  */
/* CVXGEN is Copyright (C) 2006-2017 Jacob Mattingley, jem@cvxgen.com. */
/* The code in this file is Copyright (C) 2006-2017 Jacob Mattingley. */
/* CVXGEN, or solvers produced by CVXGEN, cannot be used for commercial */
/* applications without prior written permission from Jacob Mattingley. */

/* Filename: solver.c. */
/* Description: Main solver file. */


#include <hip/hip_runtime.h>
#include <stdio.h>
#define deref(row,col,dim) row * dim + col

typedef struct Params_t {
  double Hi[15];
} Params;
typedef struct Vars_t {
  double *Ui; /* 15 rows. */
} Vars;
typedef struct Workspace_t {
  double h[15];
  double s_inv[15];
  double s_inv_z[15];
  double b[1];
  double q[15];
  double rhs[46];
  double x[46];
  double *s;
  double *z;
  double *y;
  double lhs_aff[46];
  double lhs_cc[46];
  double buffer[46];
  double buffer2[46];
  double KKT[90];
  double L[45];
  double d[46];
  double v[46];
  double d_inv[46];
  double gap;
  double optval;
  double ineq_resid_squared;
  double eq_resid_squared;
  double block_33[1];
  /* Pre-op symbols. */
  double quad_640466485248[1];
  int converged;
} Workspace;
typedef struct Settings_t {
  double resid_tol;
  double eps;
  int max_iters;
  int refine_steps;
  int better_start;
  /* Better start obviates the need for s_init and z_init. */
  double s_init;
  double z_init;
  int verbose;
  /* Show extra details of the iterative refinement steps. */
  int verbose_refinement;
  int debug;
  /* For regularization. Minimum value of abs(D_ii) in the kkt D factor. */
  double kkt_reg;
} Settings;


struct solver_scope{

    Vars vars;
    Params params;
    Workspace work;
    Settings settings;
    int id;



    __device__ solver_scope(int _id){
        id = _id;
    };

    __device__ double eval_gap(void) {
      int i;
      double gap;
      gap = 0;
      for (i = 0; i < 15; i++)
        gap += work.z[i]*work.s[i];
      return gap;
    }
    __device__ void set_defaults(void) {
      settings.resid_tol = 1e-6;
      settings.eps = 1e-4;
      settings.max_iters = 25;
      settings.refine_steps = 1;
      settings.s_init = 1;
      settings.z_init = 1;
      settings.debug = 0;
      settings.verbose = 0;
      settings.verbose_refinement = 0;
      settings.better_start = 1;
      settings.kkt_reg = 1e-7;
    }
    __device__ void setup_pointers(void) {
      work.y = work.x + 15;
      work.s = work.x + 16;
      work.z = work.x + 31;
      vars.Ui = work.x + 0;
    }
    __device__ void setup_indexing(void) {
      setup_pointers();
    }
    __device__ void set_start(void) {
      int i;
      for (i = 0; i < 15; i++)
        work.x[i] = 0;
      for (i = 0; i < 1; i++)
        work.y[i] = 0;
      for (i = 0; i < 15; i++)
        work.s[i] = (work.h[i] > 0) ? work.h[i] : settings.s_init;
      for (i = 0; i < 15; i++)
        work.z[i] = settings.z_init;
    }
    __device__ double eval_objv(void) {
      int i;
      double objv;
      /* Borrow space in work.rhs. */
      multbyP(work.rhs, work.x);
      objv = 0;
      for (i = 0; i < 15; i++)
        objv += work.x[i]*work.rhs[i];
      objv *= 0.5;
      for (i = 0; i < 15; i++)
        objv += work.q[i]*work.x[i];
      objv += work.quad_640466485248[0];
      return objv;
    }
    __device__ void fillrhs_aff(void) {
      int i;
      double *r1, *r2, *r3, *r4;
      r1 = work.rhs;
      r2 = work.rhs + 15;
      r3 = work.rhs + 30;
      r4 = work.rhs + 45;
      /* r1 = -A^Ty - G^Tz - Px - q. */
      multbymAT(r1, work.y);
      multbymGT(work.buffer, work.z);
      for (i = 0; i < 15; i++)
        r1[i] += work.buffer[i];
      multbyP(work.buffer, work.x);
      for (i = 0; i < 15; i++)
        r1[i] -= work.buffer[i] + work.q[i];
      /* r2 = -z. */
      for (i = 0; i < 15; i++)
        r2[i] = -work.z[i];
      /* r3 = -Gx - s + h. */
      multbymG(r3, work.x);
      for (i = 0; i < 15; i++)
        r3[i] += -work.s[i] + work.h[i];
      /* r4 = -Ax + b. */
      multbymA(r4, work.x);
      for (i = 0; i < 1; i++)
        r4[i] += work.b[i];
    }
    __device__ void fillrhs_cc(void) {
      int i;
      double *r2;
      double *ds_aff, *dz_aff;
      double mu;
      double alpha;
      double sigma;
      double smu;
      double minval;
      r2 = work.rhs + 15;
      ds_aff = work.lhs_aff + 15;
      dz_aff = work.lhs_aff + 30;
      mu = 0;
      for (i = 0; i < 15; i++)
        mu += work.s[i]*work.z[i];
      /* Don't finish calculating mu quite yet. */
      /* Find min(min(ds./s), min(dz./z)). */
      minval = 0;
      for (i = 0; i < 15; i++)
        if (ds_aff[i] < minval*work.s[i])
          minval = ds_aff[i]/work.s[i];
      for (i = 0; i < 15; i++)
        if (dz_aff[i] < minval*work.z[i])
          minval = dz_aff[i]/work.z[i];
      /* Find alpha. */
      if (-1 < minval)
          alpha = 1;
      else
          alpha = -1/minval;
      sigma = 0;
      for (i = 0; i < 15; i++)
        sigma += (work.s[i] + alpha*ds_aff[i])*
          (work.z[i] + alpha*dz_aff[i]);
      sigma /= mu;
      sigma = sigma*sigma*sigma;
      /* Finish calculating mu now. */
      mu *= 0.06666666666666667;
      smu = sigma*mu;
      /* Fill-in the rhs. */
      for (i = 0; i < 15; i++)
        work.rhs[i] = 0;
      for (i = 30; i < 46; i++)
        work.rhs[i] = 0;
      for (i = 0; i < 15; i++)
        r2[i] = work.s_inv[i]*(smu - ds_aff[i]*dz_aff[i]);
    }
    __device__ void refine(double *target, double *var) {
      int i, j;
      double *residual = work.buffer;
      double norm2;
      double *new_var = work.buffer2;
      for (j = 0; j < settings.refine_steps; j++) {
        norm2 = 0;
        matrix_multiply(residual, var);
        for (i = 0; i < 46; i++) {
          residual[i] = residual[i] - target[i];
          norm2 += residual[i]*residual[i];
        }
    #ifndef ZERO_LIBRARY_MODE
        if (settings.verbose_refinement) {
          if (j == 0)
            printf("Initial residual before refinement has norm squared %.6g.\n", norm2);
          else
            printf("After refinement we get squared norm %.6g.\n", norm2);
        }
    #endif
        /* Solve to find new_var = KKT \ (target - A*var). */
        ldl_solve(residual, new_var);
        /* Update var += new_var, or var += KKT \ (target - A*var). */
        for (i = 0; i < 46; i++) {
          var[i] -= new_var[i];
        }
      }
    #ifndef ZERO_LIBRARY_MODE
      if (settings.verbose_refinement) {
        /* Check the residual once more, but only if we're reporting it, since */
        /* it's expensive. */
        norm2 = 0;
        matrix_multiply(residual, var);
        for (i = 0; i < 46; i++) {
          residual[i] = residual[i] - target[i];
          norm2 += residual[i]*residual[i];
        }
        if (j == 0)
          printf("Initial residual before refinement has norm squared %.6g.\n", norm2);
        else
          printf("After refinement we get squared norm %.6g.\n", norm2);
      }
    #endif
    }
    __device__ double calc_ineq_resid_squared(void) {
      /* Calculates the norm ||-Gx - s + h||. */
      double norm2_squared;
      int i;
      /* Find -Gx. */
      multbymG(work.buffer, work.x);
      /* Add -s + h. */
      for (i = 0; i < 15; i++)
        work.buffer[i] += -work.s[i] + work.h[i];
      /* Now find the squared norm. */
      norm2_squared = 0;
      for (i = 0; i < 15; i++)
        norm2_squared += work.buffer[i]*work.buffer[i];
      return norm2_squared;
    }
    __device__ double calc_eq_resid_squared(void) {
      /* Calculates the norm ||-Ax + b||. */
      double norm2_squared;
      int i;
      /* Find -Ax. */
      multbymA(work.buffer, work.x);
      /* Add +b. */
      for (i = 0; i < 1; i++)
        work.buffer[i] += work.b[i];
      /* Now find the squared norm. */
      norm2_squared = 0;
      for (i = 0; i < 1; i++)
        norm2_squared += work.buffer[i]*work.buffer[i];
      return norm2_squared;
    }
    __device__ void better_start(void) {
      /* Calculates a better starting point, using a similar approach to CVXOPT. */
      /* Not yet speed optimized. */
      int i;
      double *x, *s, *z, *y;
      double alpha;
      work.block_33[0] = -1;
      /* Make sure sinvz is 1 to make hijacked KKT system ok. */
      for (i = 0; i < 15; i++)
        work.s_inv_z[i] = 1;
      fill_KKT();
      ldl_factor();
      fillrhs_start();
      /* Borrow work.lhs_aff for the solution. */
      ldl_solve(work.rhs, work.lhs_aff);
      /* Don't do any refinement for now. Precision doesn't matter too much. */
      x = work.lhs_aff;
      s = work.lhs_aff + 15;
      z = work.lhs_aff + 30;
      y = work.lhs_aff + 45;
      /* Just set x and y as is. */
      for (i = 0; i < 15; i++)
        work.x[i] = x[i];
      for (i = 0; i < 1; i++)
        work.y[i] = y[i];
      /* Now complete the initialization. Start with s. */
      /* Must have alpha > max(z). */
      alpha = -1e99;
      for (i = 0; i < 15; i++)
        if (alpha < z[i])
          alpha = z[i];
      if (alpha < 0) {
        for (i = 0; i < 15; i++)
          work.s[i] = -z[i];
      } else {
        alpha += 1;
        for (i = 0; i < 15; i++)
          work.s[i] = -z[i] + alpha;
      }
      /* Now initialize z. */
      /* Now must have alpha > max(-z). */
      alpha = -1e99;
      for (i = 0; i < 15; i++)
        if (alpha < -z[i])
          alpha = -z[i];
      if (alpha < 0) {
        for (i = 0; i < 15; i++)
          work.z[i] = z[i];
      } else {
        alpha += 1;
        for (i = 0; i < 15; i++)
          work.z[i] = z[i] + alpha;
      }
    }
    __device__ void fillrhs_start(void) {
      /* Fill rhs with (-q, 0, h, b). */
      int i;
      double *r1, *r2, *r3, *r4;
      r1 = work.rhs;
      r2 = work.rhs + 15;
      r3 = work.rhs + 30;
      r4 = work.rhs + 45;
      for (i = 0; i < 15; i++)
        r1[i] = -work.q[i];
      for (i = 0; i < 15; i++)
        r2[i] = 0;
      for (i = 0; i < 15; i++)
        r3[i] = work.h[i];
      for (i = 0; i < 1; i++)
        r4[i] = work.b[i];
    }
    __device__ long solve(void) {
      int i;
      int iter;
      double *dx, *ds, *dy, *dz;
      double minval;
      double alpha;
      work.converged = 0;
      setup_pointers();
      pre_ops();
    #ifndef ZERO_LIBRARY_MODE
      if (settings.verbose)
        printf("iter     objv        gap       |Ax-b|    |Gx+s-h|    step\n");
    #endif
      fillq();
      fillh();
      fillb();
      if (settings.better_start)
        better_start();
      else
        set_start();
      for (iter = 0; iter < settings.max_iters; iter++) {
        for (i = 0; i < 15; i++) {
          work.s_inv[i] = 1.0 / work.s[i];
          work.s_inv_z[i] = work.s_inv[i]*work.z[i];
        }
        work.block_33[0] = 0;
        fill_KKT();
        ldl_factor();
        /* Affine scaling directions. */
        fillrhs_aff();
        ldl_solve(work.rhs, work.lhs_aff);
        refine(work.rhs, work.lhs_aff);
        /* Centering plus corrector directions. */
        fillrhs_cc();
        ldl_solve(work.rhs, work.lhs_cc);
        refine(work.rhs, work.lhs_cc);
        /* Add the two together and store in aff. */
        for (i = 0; i < 46; i++)
          work.lhs_aff[i] += work.lhs_cc[i];
        /* Rename aff to reflect its new meaning. */
        dx = work.lhs_aff;
        ds = work.lhs_aff + 15;
        dz = work.lhs_aff + 30;
        dy = work.lhs_aff + 45;
        /* Find min(min(ds./s), min(dz./z)). */
        minval = 0;
        for (i = 0; i < 15; i++)
          if (ds[i] < minval*work.s[i])
            minval = ds[i]/work.s[i];
        for (i = 0; i < 15; i++)
          if (dz[i] < minval*work.z[i])
            minval = dz[i]/work.z[i];
        /* Find alpha. */
        if (-0.99 < minval)
          alpha = 1;
        else
          alpha = -0.99/minval;
        /* Update the primal and dual variables. */
        for (i = 0; i < 15; i++)
          work.x[i] += alpha*dx[i];
        for (i = 0; i < 15; i++)
          work.s[i] += alpha*ds[i];
        for (i = 0; i < 15; i++)
          work.z[i] += alpha*dz[i];
        for (i = 0; i < 1; i++)
          work.y[i] += alpha*dy[i];
        work.gap = eval_gap();
        work.eq_resid_squared = calc_eq_resid_squared();
        work.ineq_resid_squared = calc_ineq_resid_squared();
    #ifndef ZERO_LIBRARY_MODE
        if (settings.verbose) {
          work.optval = eval_objv();
          printf("%3d   %10.3e  %9.2e  %9.2e  %9.2e  % 6.4f\n",
              iter+1, work.optval, work.gap, sqrt(work.eq_resid_squared),
              sqrt(work.ineq_resid_squared), alpha);
        }
    #endif
        /* Test termination conditions. Requires optimality, and satisfied */
        /* constraints. */
        if (   (work.gap < settings.eps)
            && (work.eq_resid_squared <= settings.resid_tol*settings.resid_tol)
            && (work.ineq_resid_squared <= settings.resid_tol*settings.resid_tol)
           ) {
          work.converged = 1;
          work.optval = eval_objv();
          return iter+1;
        }
      }
      return iter;
    }



    __device__ void multbymA(double *lhs, double *rhs) {
      lhs[0] = -rhs[0]*(1)-rhs[1]*(1)-rhs[2]*(1)-rhs[3]*(1)-rhs[4]*(1)-rhs[5]*(1)-rhs[6]*(1)-rhs[7]*(1)-rhs[8]*(1)-rhs[9]*(1)-rhs[10]*(1)-rhs[11]*(1)-rhs[12]*(1)-rhs[13]*(1)-rhs[14]*(1);
    }
    __device__ void multbymAT(double *lhs, double *rhs) {
      lhs[0] = -rhs[0]*(1);
      lhs[1] = -rhs[0]*(1);
      lhs[2] = -rhs[0]*(1);
      lhs[3] = -rhs[0]*(1);
      lhs[4] = -rhs[0]*(1);
      lhs[5] = -rhs[0]*(1);
      lhs[6] = -rhs[0]*(1);
      lhs[7] = -rhs[0]*(1);
      lhs[8] = -rhs[0]*(1);
      lhs[9] = -rhs[0]*(1);
      lhs[10] = -rhs[0]*(1);
      lhs[11] = -rhs[0]*(1);
      lhs[12] = -rhs[0]*(1);
      lhs[13] = -rhs[0]*(1);
      lhs[14] = -rhs[0]*(1);
    }
    __device__ void multbymG(double *lhs, double *rhs) {
      lhs[0] = -rhs[0]*(-1);
      lhs[1] = -rhs[1]*(-1);
      lhs[2] = -rhs[2]*(-1);
      lhs[3] = -rhs[3]*(-1);
      lhs[4] = -rhs[4]*(-1);
      lhs[5] = -rhs[5]*(-1);
      lhs[6] = -rhs[6]*(-1);
      lhs[7] = -rhs[7]*(-1);
      lhs[8] = -rhs[8]*(-1);
      lhs[9] = -rhs[9]*(-1);
      lhs[10] = -rhs[10]*(-1);
      lhs[11] = -rhs[11]*(-1);
      lhs[12] = -rhs[12]*(-1);
      lhs[13] = -rhs[13]*(-1);
      lhs[14] = -rhs[14]*(-1);
    }
    __device__ void multbymGT(double *lhs, double *rhs) {
      lhs[0] = -rhs[0]*(-1);
      lhs[1] = -rhs[1]*(-1);
      lhs[2] = -rhs[2]*(-1);
      lhs[3] = -rhs[3]*(-1);
      lhs[4] = -rhs[4]*(-1);
      lhs[5] = -rhs[5]*(-1);
      lhs[6] = -rhs[6]*(-1);
      lhs[7] = -rhs[7]*(-1);
      lhs[8] = -rhs[8]*(-1);
      lhs[9] = -rhs[9]*(-1);
      lhs[10] = -rhs[10]*(-1);
      lhs[11] = -rhs[11]*(-1);
      lhs[12] = -rhs[12]*(-1);
      lhs[13] = -rhs[13]*(-1);
      lhs[14] = -rhs[14]*(-1);
    }
    __device__ void multbyP(double *lhs, double *rhs) {
      /* TODO use the fact that P is symmetric? */
      /* TODO check doubling / half factor etc. */
      lhs[0] = rhs[0]*(2);
      lhs[1] = rhs[1]*(2);
      lhs[2] = rhs[2]*(2);
      lhs[3] = rhs[3]*(2);
      lhs[4] = rhs[4]*(2);
      lhs[5] = rhs[5]*(2);
      lhs[6] = rhs[6]*(2);
      lhs[7] = rhs[7]*(2);
      lhs[8] = rhs[8]*(2);
      lhs[9] = rhs[9]*(2);
      lhs[10] = rhs[10]*(2);
      lhs[11] = rhs[11]*(2);
      lhs[12] = rhs[12]*(2);
      lhs[13] = rhs[13]*(2);
      lhs[14] = rhs[14]*(2);
    }
    __device__ void fillq(void) {
      work.q[0] = -2*params.Hi[0];
      work.q[1] = -2*params.Hi[1];
      work.q[2] = -2*params.Hi[2];
      work.q[3] = -2*params.Hi[3];
      work.q[4] = -2*params.Hi[4];
      work.q[5] = -2*params.Hi[5];
      work.q[6] = -2*params.Hi[6];
      work.q[7] = -2*params.Hi[7];
      work.q[8] = -2*params.Hi[8];
      work.q[9] = -2*params.Hi[9];
      work.q[10] = -2*params.Hi[10];
      work.q[11] = -2*params.Hi[11];
      work.q[12] = -2*params.Hi[12];
      work.q[13] = -2*params.Hi[13];
      work.q[14] = -2*params.Hi[14];
    }
    __device__ void fillh(void) {
      work.h[0] = 0;
      work.h[1] = 0;
      work.h[2] = 0;
      work.h[3] = 0;
      work.h[4] = 0;
      work.h[5] = 0;
      work.h[6] = 0;
      work.h[7] = 0;
      work.h[8] = 0;
      work.h[9] = 0;
      work.h[10] = 0;
      work.h[11] = 0;
      work.h[12] = 0;
      work.h[13] = 0;
      work.h[14] = 0;
    }
    __device__ void fillb(void) {
      work.b[0] = 1;
    }
    __device__ void pre_ops(void) {
      work.quad_640466485248[0] = params.Hi[0]*params.Hi[0]+params.Hi[1]*params.Hi[1]+params.Hi[2]*params.Hi[2]+params.Hi[3]*params.Hi[3]+params.Hi[4]*params.Hi[4]+params.Hi[5]*params.Hi[5]+params.Hi[6]*params.Hi[6]+params.Hi[7]*params.Hi[7]+params.Hi[8]*params.Hi[8]+params.Hi[9]*params.Hi[9]+params.Hi[10]*params.Hi[10]+params.Hi[11]*params.Hi[11]+params.Hi[12]*params.Hi[12]+params.Hi[13]*params.Hi[13]+params.Hi[14]*params.Hi[14];
    }


    /* Be sure to place ldl_solve first, so storage schemes are defined by it. */
    __device__ void ldl_solve(double *target, double *var) {
      int i;
      /* Find var = (L*diag(work.d)*L') \ target, then unpermute. */
      /* Answer goes into var. */
      /* Forward substitution. */
      /* Include permutation as we retrieve from target. Use v so we can unpermute */
      /* later. */
      work.v[0] = target[15];
      work.v[1] = target[16];
      work.v[2] = target[17];
      work.v[3] = target[18];
      work.v[4] = target[19];
      work.v[5] = target[20];
      work.v[6] = target[21];
      work.v[7] = target[22];
      work.v[8] = target[23];
      work.v[9] = target[24];
      work.v[10] = target[25];
      work.v[11] = target[26];
      work.v[12] = target[27];
      work.v[13] = target[28];
      work.v[14] = target[29];
      work.v[15] = target[30]-work.L[0]*work.v[0];
      work.v[16] = target[31]-work.L[1]*work.v[1];
      work.v[17] = target[32]-work.L[2]*work.v[2];
      work.v[18] = target[33]-work.L[3]*work.v[3];
      work.v[19] = target[34]-work.L[4]*work.v[4];
      work.v[20] = target[35]-work.L[5]*work.v[5];
      work.v[21] = target[36]-work.L[6]*work.v[6];
      work.v[22] = target[37]-work.L[7]*work.v[7];
      work.v[23] = target[38]-work.L[8]*work.v[8];
      work.v[24] = target[39]-work.L[9]*work.v[9];
      work.v[25] = target[40]-work.L[10]*work.v[10];
      work.v[26] = target[41]-work.L[11]*work.v[11];
      work.v[27] = target[42]-work.L[12]*work.v[12];
      work.v[28] = target[43]-work.L[13]*work.v[13];
      work.v[29] = target[44]-work.L[14]*work.v[14];
      work.v[30] = target[0]-work.L[15]*work.v[15];
      work.v[31] = target[1]-work.L[16]*work.v[16];
      work.v[32] = target[2]-work.L[17]*work.v[17];
      work.v[33] = target[3]-work.L[18]*work.v[18];
      work.v[34] = target[4]-work.L[19]*work.v[19];
      work.v[35] = target[5]-work.L[20]*work.v[20];
      work.v[36] = target[6]-work.L[21]*work.v[21];
      work.v[37] = target[7]-work.L[22]*work.v[22];
      work.v[38] = target[8]-work.L[23]*work.v[23];
      work.v[39] = target[9]-work.L[24]*work.v[24];
      work.v[40] = target[10]-work.L[25]*work.v[25];
      work.v[41] = target[11]-work.L[26]*work.v[26];
      work.v[42] = target[12]-work.L[27]*work.v[27];
      work.v[43] = target[13]-work.L[28]*work.v[28];
      work.v[44] = target[14]-work.L[29]*work.v[29];
      work.v[45] = target[45]-work.L[30]*work.v[30]-work.L[31]*work.v[31]-work.L[32]*work.v[32]-work.L[33]*work.v[33]-work.L[34]*work.v[34]-work.L[35]*work.v[35]-work.L[36]*work.v[36]-work.L[37]*work.v[37]-work.L[38]*work.v[38]-work.L[39]*work.v[39]-work.L[40]*work.v[40]-work.L[41]*work.v[41]-work.L[42]*work.v[42]-work.L[43]*work.v[43]-work.L[44]*work.v[44];
      /* Diagonal scaling. Assume correctness of work.d_inv. */
      for (i = 0; i < 46; i++)
        work.v[i] *= work.d_inv[i];
      /* Back substitution */
      work.v[44] -= work.L[44]*work.v[45];
      work.v[43] -= work.L[43]*work.v[45];
      work.v[42] -= work.L[42]*work.v[45];
      work.v[41] -= work.L[41]*work.v[45];
      work.v[40] -= work.L[40]*work.v[45];
      work.v[39] -= work.L[39]*work.v[45];
      work.v[38] -= work.L[38]*work.v[45];
      work.v[37] -= work.L[37]*work.v[45];
      work.v[36] -= work.L[36]*work.v[45];
      work.v[35] -= work.L[35]*work.v[45];
      work.v[34] -= work.L[34]*work.v[45];
      work.v[33] -= work.L[33]*work.v[45];
      work.v[32] -= work.L[32]*work.v[45];
      work.v[31] -= work.L[31]*work.v[45];
      work.v[30] -= work.L[30]*work.v[45];
      work.v[29] -= work.L[29]*work.v[44];
      work.v[28] -= work.L[28]*work.v[43];
      work.v[27] -= work.L[27]*work.v[42];
      work.v[26] -= work.L[26]*work.v[41];
      work.v[25] -= work.L[25]*work.v[40];
      work.v[24] -= work.L[24]*work.v[39];
      work.v[23] -= work.L[23]*work.v[38];
      work.v[22] -= work.L[22]*work.v[37];
      work.v[21] -= work.L[21]*work.v[36];
      work.v[20] -= work.L[20]*work.v[35];
      work.v[19] -= work.L[19]*work.v[34];
      work.v[18] -= work.L[18]*work.v[33];
      work.v[17] -= work.L[17]*work.v[32];
      work.v[16] -= work.L[16]*work.v[31];
      work.v[15] -= work.L[15]*work.v[30];
      work.v[14] -= work.L[14]*work.v[29];
      work.v[13] -= work.L[13]*work.v[28];
      work.v[12] -= work.L[12]*work.v[27];
      work.v[11] -= work.L[11]*work.v[26];
      work.v[10] -= work.L[10]*work.v[25];
      work.v[9] -= work.L[9]*work.v[24];
      work.v[8] -= work.L[8]*work.v[23];
      work.v[7] -= work.L[7]*work.v[22];
      work.v[6] -= work.L[6]*work.v[21];
      work.v[5] -= work.L[5]*work.v[20];
      work.v[4] -= work.L[4]*work.v[19];
      work.v[3] -= work.L[3]*work.v[18];
      work.v[2] -= work.L[2]*work.v[17];
      work.v[1] -= work.L[1]*work.v[16];
      work.v[0] -= work.L[0]*work.v[15];
      /* Unpermute the result, from v to var. */
      var[0] = work.v[30];
      var[1] = work.v[31];
      var[2] = work.v[32];
      var[3] = work.v[33];
      var[4] = work.v[34];
      var[5] = work.v[35];
      var[6] = work.v[36];
      var[7] = work.v[37];
      var[8] = work.v[38];
      var[9] = work.v[39];
      var[10] = work.v[40];
      var[11] = work.v[41];
      var[12] = work.v[42];
      var[13] = work.v[43];
      var[14] = work.v[44];
      var[15] = work.v[0];
      var[16] = work.v[1];
      var[17] = work.v[2];
      var[18] = work.v[3];
      var[19] = work.v[4];
      var[20] = work.v[5];
      var[21] = work.v[6];
      var[22] = work.v[7];
      var[23] = work.v[8];
      var[24] = work.v[9];
      var[25] = work.v[10];
      var[26] = work.v[11];
      var[27] = work.v[12];
      var[28] = work.v[13];
      var[29] = work.v[14];
      var[30] = work.v[15];
      var[31] = work.v[16];
      var[32] = work.v[17];
      var[33] = work.v[18];
      var[34] = work.v[19];
      var[35] = work.v[20];
      var[36] = work.v[21];
      var[37] = work.v[22];
      var[38] = work.v[23];
      var[39] = work.v[24];
      var[40] = work.v[25];
      var[41] = work.v[26];
      var[42] = work.v[27];
      var[43] = work.v[28];
      var[44] = work.v[29];
      var[45] = work.v[45];
    #ifndef ZERO_LIBRARY_MODE
      if (settings.debug) {
        printf("Squared norm for solution is %.8g.\n", check_residual(target, var));
      }
    #endif
    }
    __device__ void ldl_factor(void) {
      work.d[0] = work.KKT[0];
      if (work.d[0] < 0)
        work.d[0] = settings.kkt_reg;
      else
        work.d[0] += settings.kkt_reg;
      work.d_inv[0] = 1/work.d[0];
      work.L[0] = work.KKT[1]*work.d_inv[0];
      work.v[1] = work.KKT[2];
      work.d[1] = work.v[1];
      if (work.d[1] < 0)
        work.d[1] = settings.kkt_reg;
      else
        work.d[1] += settings.kkt_reg;
      work.d_inv[1] = 1/work.d[1];
      work.L[1] = (work.KKT[3])*work.d_inv[1];
      work.v[2] = work.KKT[4];
      work.d[2] = work.v[2];
      if (work.d[2] < 0)
        work.d[2] = settings.kkt_reg;
      else
        work.d[2] += settings.kkt_reg;
      work.d_inv[2] = 1/work.d[2];
      work.L[2] = (work.KKT[5])*work.d_inv[2];
      work.v[3] = work.KKT[6];
      work.d[3] = work.v[3];
      if (work.d[3] < 0)
        work.d[3] = settings.kkt_reg;
      else
        work.d[3] += settings.kkt_reg;
      work.d_inv[3] = 1/work.d[3];
      work.L[3] = (work.KKT[7])*work.d_inv[3];
      work.v[4] = work.KKT[8];
      work.d[4] = work.v[4];
      if (work.d[4] < 0)
        work.d[4] = settings.kkt_reg;
      else
        work.d[4] += settings.kkt_reg;
      work.d_inv[4] = 1/work.d[4];
      work.L[4] = (work.KKT[9])*work.d_inv[4];
      work.v[5] = work.KKT[10];
      work.d[5] = work.v[5];
      if (work.d[5] < 0)
        work.d[5] = settings.kkt_reg;
      else
        work.d[5] += settings.kkt_reg;
      work.d_inv[5] = 1/work.d[5];
      work.L[5] = (work.KKT[11])*work.d_inv[5];
      work.v[6] = work.KKT[12];
      work.d[6] = work.v[6];
      if (work.d[6] < 0)
        work.d[6] = settings.kkt_reg;
      else
        work.d[6] += settings.kkt_reg;
      work.d_inv[6] = 1/work.d[6];
      work.L[6] = (work.KKT[13])*work.d_inv[6];
      work.v[7] = work.KKT[14];
      work.d[7] = work.v[7];
      if (work.d[7] < 0)
        work.d[7] = settings.kkt_reg;
      else
        work.d[7] += settings.kkt_reg;
      work.d_inv[7] = 1/work.d[7];
      work.L[7] = (work.KKT[15])*work.d_inv[7];
      work.v[8] = work.KKT[16];
      work.d[8] = work.v[8];
      if (work.d[8] < 0)
        work.d[8] = settings.kkt_reg;
      else
        work.d[8] += settings.kkt_reg;
      work.d_inv[8] = 1/work.d[8];
      work.L[8] = (work.KKT[17])*work.d_inv[8];
      work.v[9] = work.KKT[18];
      work.d[9] = work.v[9];
      if (work.d[9] < 0)
        work.d[9] = settings.kkt_reg;
      else
        work.d[9] += settings.kkt_reg;
      work.d_inv[9] = 1/work.d[9];
      work.L[9] = (work.KKT[19])*work.d_inv[9];
      work.v[10] = work.KKT[20];
      work.d[10] = work.v[10];
      if (work.d[10] < 0)
        work.d[10] = settings.kkt_reg;
      else
        work.d[10] += settings.kkt_reg;
      work.d_inv[10] = 1/work.d[10];
      work.L[10] = (work.KKT[21])*work.d_inv[10];
      work.v[11] = work.KKT[22];
      work.d[11] = work.v[11];
      if (work.d[11] < 0)
        work.d[11] = settings.kkt_reg;
      else
        work.d[11] += settings.kkt_reg;
      work.d_inv[11] = 1/work.d[11];
      work.L[11] = (work.KKT[23])*work.d_inv[11];
      work.v[12] = work.KKT[24];
      work.d[12] = work.v[12];
      if (work.d[12] < 0)
        work.d[12] = settings.kkt_reg;
      else
        work.d[12] += settings.kkt_reg;
      work.d_inv[12] = 1/work.d[12];
      work.L[12] = (work.KKT[25])*work.d_inv[12];
      work.v[13] = work.KKT[26];
      work.d[13] = work.v[13];
      if (work.d[13] < 0)
        work.d[13] = settings.kkt_reg;
      else
        work.d[13] += settings.kkt_reg;
      work.d_inv[13] = 1/work.d[13];
      work.L[13] = (work.KKT[27])*work.d_inv[13];
      work.v[14] = work.KKT[28];
      work.d[14] = work.v[14];
      if (work.d[14] < 0)
        work.d[14] = settings.kkt_reg;
      else
        work.d[14] += settings.kkt_reg;
      work.d_inv[14] = 1/work.d[14];
      work.L[14] = (work.KKT[29])*work.d_inv[14];
      work.v[0] = work.L[0]*work.d[0];
      work.v[15] = work.KKT[30]-work.L[0]*work.v[0];
      work.d[15] = work.v[15];
      if (work.d[15] > 0)
        work.d[15] = -settings.kkt_reg;
      else
        work.d[15] -= settings.kkt_reg;
      work.d_inv[15] = 1/work.d[15];
      work.L[15] = (work.KKT[31])*work.d_inv[15];
      work.v[1] = work.L[1]*work.d[1];
      work.v[16] = work.KKT[32]-work.L[1]*work.v[1];
      work.d[16] = work.v[16];
      if (work.d[16] > 0)
        work.d[16] = -settings.kkt_reg;
      else
        work.d[16] -= settings.kkt_reg;
      work.d_inv[16] = 1/work.d[16];
      work.L[16] = (work.KKT[33])*work.d_inv[16];
      work.v[2] = work.L[2]*work.d[2];
      work.v[17] = work.KKT[34]-work.L[2]*work.v[2];
      work.d[17] = work.v[17];
      if (work.d[17] > 0)
        work.d[17] = -settings.kkt_reg;
      else
        work.d[17] -= settings.kkt_reg;
      work.d_inv[17] = 1/work.d[17];
      work.L[17] = (work.KKT[35])*work.d_inv[17];
      work.v[3] = work.L[3]*work.d[3];
      work.v[18] = work.KKT[36]-work.L[3]*work.v[3];
      work.d[18] = work.v[18];
      if (work.d[18] > 0)
        work.d[18] = -settings.kkt_reg;
      else
        work.d[18] -= settings.kkt_reg;
      work.d_inv[18] = 1/work.d[18];
      work.L[18] = (work.KKT[37])*work.d_inv[18];
      work.v[4] = work.L[4]*work.d[4];
      work.v[19] = work.KKT[38]-work.L[4]*work.v[4];
      work.d[19] = work.v[19];
      if (work.d[19] > 0)
        work.d[19] = -settings.kkt_reg;
      else
        work.d[19] -= settings.kkt_reg;
      work.d_inv[19] = 1/work.d[19];
      work.L[19] = (work.KKT[39])*work.d_inv[19];
      work.v[5] = work.L[5]*work.d[5];
      work.v[20] = work.KKT[40]-work.L[5]*work.v[5];
      work.d[20] = work.v[20];
      if (work.d[20] > 0)
        work.d[20] = -settings.kkt_reg;
      else
        work.d[20] -= settings.kkt_reg;
      work.d_inv[20] = 1/work.d[20];
      work.L[20] = (work.KKT[41])*work.d_inv[20];
      work.v[6] = work.L[6]*work.d[6];
      work.v[21] = work.KKT[42]-work.L[6]*work.v[6];
      work.d[21] = work.v[21];
      if (work.d[21] > 0)
        work.d[21] = -settings.kkt_reg;
      else
        work.d[21] -= settings.kkt_reg;
      work.d_inv[21] = 1/work.d[21];
      work.L[21] = (work.KKT[43])*work.d_inv[21];
      work.v[7] = work.L[7]*work.d[7];
      work.v[22] = work.KKT[44]-work.L[7]*work.v[7];
      work.d[22] = work.v[22];
      if (work.d[22] > 0)
        work.d[22] = -settings.kkt_reg;
      else
        work.d[22] -= settings.kkt_reg;
      work.d_inv[22] = 1/work.d[22];
      work.L[22] = (work.KKT[45])*work.d_inv[22];
      work.v[8] = work.L[8]*work.d[8];
      work.v[23] = work.KKT[46]-work.L[8]*work.v[8];
      work.d[23] = work.v[23];
      if (work.d[23] > 0)
        work.d[23] = -settings.kkt_reg;
      else
        work.d[23] -= settings.kkt_reg;
      work.d_inv[23] = 1/work.d[23];
      work.L[23] = (work.KKT[47])*work.d_inv[23];
      work.v[9] = work.L[9]*work.d[9];
      work.v[24] = work.KKT[48]-work.L[9]*work.v[9];
      work.d[24] = work.v[24];
      if (work.d[24] > 0)
        work.d[24] = -settings.kkt_reg;
      else
        work.d[24] -= settings.kkt_reg;
      work.d_inv[24] = 1/work.d[24];
      work.L[24] = (work.KKT[49])*work.d_inv[24];
      work.v[10] = work.L[10]*work.d[10];
      work.v[25] = work.KKT[50]-work.L[10]*work.v[10];
      work.d[25] = work.v[25];
      if (work.d[25] > 0)
        work.d[25] = -settings.kkt_reg;
      else
        work.d[25] -= settings.kkt_reg;
      work.d_inv[25] = 1/work.d[25];
      work.L[25] = (work.KKT[51])*work.d_inv[25];
      work.v[11] = work.L[11]*work.d[11];
      work.v[26] = work.KKT[52]-work.L[11]*work.v[11];
      work.d[26] = work.v[26];
      if (work.d[26] > 0)
        work.d[26] = -settings.kkt_reg;
      else
        work.d[26] -= settings.kkt_reg;
      work.d_inv[26] = 1/work.d[26];
      work.L[26] = (work.KKT[53])*work.d_inv[26];
      work.v[12] = work.L[12]*work.d[12];
      work.v[27] = work.KKT[54]-work.L[12]*work.v[12];
      work.d[27] = work.v[27];
      if (work.d[27] > 0)
        work.d[27] = -settings.kkt_reg;
      else
        work.d[27] -= settings.kkt_reg;
      work.d_inv[27] = 1/work.d[27];
      work.L[27] = (work.KKT[55])*work.d_inv[27];
      work.v[13] = work.L[13]*work.d[13];
      work.v[28] = work.KKT[56]-work.L[13]*work.v[13];
      work.d[28] = work.v[28];
      if (work.d[28] > 0)
        work.d[28] = -settings.kkt_reg;
      else
        work.d[28] -= settings.kkt_reg;
      work.d_inv[28] = 1/work.d[28];
      work.L[28] = (work.KKT[57])*work.d_inv[28];
      work.v[14] = work.L[14]*work.d[14];
      work.v[29] = work.KKT[58]-work.L[14]*work.v[14];
      work.d[29] = work.v[29];
      if (work.d[29] > 0)
        work.d[29] = -settings.kkt_reg;
      else
        work.d[29] -= settings.kkt_reg;
      work.d_inv[29] = 1/work.d[29];
      work.L[29] = (work.KKT[59])*work.d_inv[29];
      work.v[15] = work.L[15]*work.d[15];
      work.v[30] = work.KKT[60]-work.L[15]*work.v[15];
      work.d[30] = work.v[30];
      if (work.d[30] < 0)
        work.d[30] = settings.kkt_reg;
      else
        work.d[30] += settings.kkt_reg;
      work.d_inv[30] = 1/work.d[30];
      work.L[30] = (work.KKT[61])*work.d_inv[30];
      work.v[16] = work.L[16]*work.d[16];
      work.v[31] = work.KKT[62]-work.L[16]*work.v[16];
      work.d[31] = work.v[31];
      if (work.d[31] < 0)
        work.d[31] = settings.kkt_reg;
      else
        work.d[31] += settings.kkt_reg;
      work.d_inv[31] = 1/work.d[31];
      work.L[31] = (work.KKT[63])*work.d_inv[31];
      work.v[17] = work.L[17]*work.d[17];
      work.v[32] = work.KKT[64]-work.L[17]*work.v[17];
      work.d[32] = work.v[32];
      if (work.d[32] < 0)
        work.d[32] = settings.kkt_reg;
      else
        work.d[32] += settings.kkt_reg;
      work.d_inv[32] = 1/work.d[32];
      work.L[32] = (work.KKT[65])*work.d_inv[32];
      work.v[18] = work.L[18]*work.d[18];
      work.v[33] = work.KKT[66]-work.L[18]*work.v[18];
      work.d[33] = work.v[33];
      if (work.d[33] < 0)
        work.d[33] = settings.kkt_reg;
      else
        work.d[33] += settings.kkt_reg;
      work.d_inv[33] = 1/work.d[33];
      work.L[33] = (work.KKT[67])*work.d_inv[33];
      work.v[19] = work.L[19]*work.d[19];
      work.v[34] = work.KKT[68]-work.L[19]*work.v[19];
      work.d[34] = work.v[34];
      if (work.d[34] < 0)
        work.d[34] = settings.kkt_reg;
      else
        work.d[34] += settings.kkt_reg;
      work.d_inv[34] = 1/work.d[34];
      work.L[34] = (work.KKT[69])*work.d_inv[34];
      work.v[20] = work.L[20]*work.d[20];
      work.v[35] = work.KKT[70]-work.L[20]*work.v[20];
      work.d[35] = work.v[35];
      if (work.d[35] < 0)
        work.d[35] = settings.kkt_reg;
      else
        work.d[35] += settings.kkt_reg;
      work.d_inv[35] = 1/work.d[35];
      work.L[35] = (work.KKT[71])*work.d_inv[35];
      work.v[21] = work.L[21]*work.d[21];
      work.v[36] = work.KKT[72]-work.L[21]*work.v[21];
      work.d[36] = work.v[36];
      if (work.d[36] < 0)
        work.d[36] = settings.kkt_reg;
      else
        work.d[36] += settings.kkt_reg;
      work.d_inv[36] = 1/work.d[36];
      work.L[36] = (work.KKT[73])*work.d_inv[36];
      work.v[22] = work.L[22]*work.d[22];
      work.v[37] = work.KKT[74]-work.L[22]*work.v[22];
      work.d[37] = work.v[37];
      if (work.d[37] < 0)
        work.d[37] = settings.kkt_reg;
      else
        work.d[37] += settings.kkt_reg;
      work.d_inv[37] = 1/work.d[37];
      work.L[37] = (work.KKT[75])*work.d_inv[37];
      work.v[23] = work.L[23]*work.d[23];
      work.v[38] = work.KKT[76]-work.L[23]*work.v[23];
      work.d[38] = work.v[38];
      if (work.d[38] < 0)
        work.d[38] = settings.kkt_reg;
      else
        work.d[38] += settings.kkt_reg;
      work.d_inv[38] = 1/work.d[38];
      work.L[38] = (work.KKT[77])*work.d_inv[38];
      work.v[24] = work.L[24]*work.d[24];
      work.v[39] = work.KKT[78]-work.L[24]*work.v[24];
      work.d[39] = work.v[39];
      if (work.d[39] < 0)
        work.d[39] = settings.kkt_reg;
      else
        work.d[39] += settings.kkt_reg;
      work.d_inv[39] = 1/work.d[39];
      work.L[39] = (work.KKT[79])*work.d_inv[39];
      work.v[25] = work.L[25]*work.d[25];
      work.v[40] = work.KKT[80]-work.L[25]*work.v[25];
      work.d[40] = work.v[40];
      if (work.d[40] < 0)
        work.d[40] = settings.kkt_reg;
      else
        work.d[40] += settings.kkt_reg;
      work.d_inv[40] = 1/work.d[40];
      work.L[40] = (work.KKT[81])*work.d_inv[40];
      work.v[26] = work.L[26]*work.d[26];
      work.v[41] = work.KKT[82]-work.L[26]*work.v[26];
      work.d[41] = work.v[41];
      if (work.d[41] < 0)
        work.d[41] = settings.kkt_reg;
      else
        work.d[41] += settings.kkt_reg;
      work.d_inv[41] = 1/work.d[41];
      work.L[41] = (work.KKT[83])*work.d_inv[41];
      work.v[27] = work.L[27]*work.d[27];
      work.v[42] = work.KKT[84]-work.L[27]*work.v[27];
      work.d[42] = work.v[42];
      if (work.d[42] < 0)
        work.d[42] = settings.kkt_reg;
      else
        work.d[42] += settings.kkt_reg;
      work.d_inv[42] = 1/work.d[42];
      work.L[42] = (work.KKT[85])*work.d_inv[42];
      work.v[28] = work.L[28]*work.d[28];
      work.v[43] = work.KKT[86]-work.L[28]*work.v[28];
      work.d[43] = work.v[43];
      if (work.d[43] < 0)
        work.d[43] = settings.kkt_reg;
      else
        work.d[43] += settings.kkt_reg;
      work.d_inv[43] = 1/work.d[43];
      work.L[43] = (work.KKT[87])*work.d_inv[43];
      work.v[29] = work.L[29]*work.d[29];
      work.v[44] = work.KKT[88]-work.L[29]*work.v[29];
      work.d[44] = work.v[44];
      if (work.d[44] < 0)
        work.d[44] = settings.kkt_reg;
      else
        work.d[44] += settings.kkt_reg;
      work.d_inv[44] = 1/work.d[44];
      work.L[44] = (work.KKT[89])*work.d_inv[44];
      work.v[30] = work.L[30]*work.d[30];
      work.v[31] = work.L[31]*work.d[31];
      work.v[32] = work.L[32]*work.d[32];
      work.v[33] = work.L[33]*work.d[33];
      work.v[34] = work.L[34]*work.d[34];
      work.v[35] = work.L[35]*work.d[35];
      work.v[36] = work.L[36]*work.d[36];
      work.v[37] = work.L[37]*work.d[37];
      work.v[38] = work.L[38]*work.d[38];
      work.v[39] = work.L[39]*work.d[39];
      work.v[40] = work.L[40]*work.d[40];
      work.v[41] = work.L[41]*work.d[41];
      work.v[42] = work.L[42]*work.d[42];
      work.v[43] = work.L[43]*work.d[43];
      work.v[44] = work.L[44]*work.d[44];
      work.v[45] = 0-work.L[30]*work.v[30]-work.L[31]*work.v[31]-work.L[32]*work.v[32]-work.L[33]*work.v[33]-work.L[34]*work.v[34]-work.L[35]*work.v[35]-work.L[36]*work.v[36]-work.L[37]*work.v[37]-work.L[38]*work.v[38]-work.L[39]*work.v[39]-work.L[40]*work.v[40]-work.L[41]*work.v[41]-work.L[42]*work.v[42]-work.L[43]*work.v[43]-work.L[44]*work.v[44];
      work.d[45] = work.v[45];
      if (work.d[45] > 0)
        work.d[45] = -settings.kkt_reg;
      else
        work.d[45] -= settings.kkt_reg;
      work.d_inv[45] = 1/work.d[45];
    #ifndef ZERO_LIBRARY_MODE
      if (settings.debug) {
        printf("Squared Frobenius for factorization is %.8g.\n", check_factorization());
      }
    #endif
    }
    __device__ double check_factorization(void) {
      /* Returns the squared Frobenius norm of A - L*D*L'. */
      double temp, residual;
      /* Only check the lower triangle. */
      residual = 0;
      temp = work.KKT[60]-1*work.d[30]*1-work.L[15]*work.d[15]*work.L[15];
      residual += temp*temp;
      temp = work.KKT[62]-1*work.d[31]*1-work.L[16]*work.d[16]*work.L[16];
      residual += temp*temp;
      temp = work.KKT[64]-1*work.d[32]*1-work.L[17]*work.d[17]*work.L[17];
      residual += temp*temp;
      temp = work.KKT[66]-1*work.d[33]*1-work.L[18]*work.d[18]*work.L[18];
      residual += temp*temp;
      temp = work.KKT[68]-1*work.d[34]*1-work.L[19]*work.d[19]*work.L[19];
      residual += temp*temp;
      temp = work.KKT[70]-1*work.d[35]*1-work.L[20]*work.d[20]*work.L[20];
      residual += temp*temp;
      temp = work.KKT[72]-1*work.d[36]*1-work.L[21]*work.d[21]*work.L[21];
      residual += temp*temp;
      temp = work.KKT[74]-1*work.d[37]*1-work.L[22]*work.d[22]*work.L[22];
      residual += temp*temp;
      temp = work.KKT[76]-1*work.d[38]*1-work.L[23]*work.d[23]*work.L[23];
      residual += temp*temp;
      temp = work.KKT[78]-1*work.d[39]*1-work.L[24]*work.d[24]*work.L[24];
      residual += temp*temp;
      temp = work.KKT[80]-1*work.d[40]*1-work.L[25]*work.d[25]*work.L[25];
      residual += temp*temp;
      temp = work.KKT[82]-1*work.d[41]*1-work.L[26]*work.d[26]*work.L[26];
      residual += temp*temp;
      temp = work.KKT[84]-1*work.d[42]*1-work.L[27]*work.d[27]*work.L[27];
      residual += temp*temp;
      temp = work.KKT[86]-1*work.d[43]*1-work.L[28]*work.d[28]*work.L[28];
      residual += temp*temp;
      temp = work.KKT[88]-1*work.d[44]*1-work.L[29]*work.d[29]*work.L[29];
      residual += temp*temp;
      temp = work.KKT[0]-1*work.d[0]*1;
      residual += temp*temp;
      temp = work.KKT[2]-1*work.d[1]*1;
      residual += temp*temp;
      temp = work.KKT[4]-1*work.d[2]*1;
      residual += temp*temp;
      temp = work.KKT[6]-1*work.d[3]*1;
      residual += temp*temp;
      temp = work.KKT[8]-1*work.d[4]*1;
      residual += temp*temp;
      temp = work.KKT[10]-1*work.d[5]*1;
      residual += temp*temp;
      temp = work.KKT[12]-1*work.d[6]*1;
      residual += temp*temp;
      temp = work.KKT[14]-1*work.d[7]*1;
      residual += temp*temp;
      temp = work.KKT[16]-1*work.d[8]*1;
      residual += temp*temp;
      temp = work.KKT[18]-1*work.d[9]*1;
      residual += temp*temp;
      temp = work.KKT[20]-1*work.d[10]*1;
      residual += temp*temp;
      temp = work.KKT[22]-1*work.d[11]*1;
      residual += temp*temp;
      temp = work.KKT[24]-1*work.d[12]*1;
      residual += temp*temp;
      temp = work.KKT[26]-1*work.d[13]*1;
      residual += temp*temp;
      temp = work.KKT[28]-1*work.d[14]*1;
      residual += temp*temp;
      temp = work.KKT[1]-work.L[0]*work.d[0]*1;
      residual += temp*temp;
      temp = work.KKT[3]-work.L[1]*work.d[1]*1;
      residual += temp*temp;
      temp = work.KKT[5]-work.L[2]*work.d[2]*1;
      residual += temp*temp;
      temp = work.KKT[7]-work.L[3]*work.d[3]*1;
      residual += temp*temp;
      temp = work.KKT[9]-work.L[4]*work.d[4]*1;
      residual += temp*temp;
      temp = work.KKT[11]-work.L[5]*work.d[5]*1;
      residual += temp*temp;
      temp = work.KKT[13]-work.L[6]*work.d[6]*1;
      residual += temp*temp;
      temp = work.KKT[15]-work.L[7]*work.d[7]*1;
      residual += temp*temp;
      temp = work.KKT[17]-work.L[8]*work.d[8]*1;
      residual += temp*temp;
      temp = work.KKT[19]-work.L[9]*work.d[9]*1;
      residual += temp*temp;
      temp = work.KKT[21]-work.L[10]*work.d[10]*1;
      residual += temp*temp;
      temp = work.KKT[23]-work.L[11]*work.d[11]*1;
      residual += temp*temp;
      temp = work.KKT[25]-work.L[12]*work.d[12]*1;
      residual += temp*temp;
      temp = work.KKT[27]-work.L[13]*work.d[13]*1;
      residual += temp*temp;
      temp = work.KKT[29]-work.L[14]*work.d[14]*1;
      residual += temp*temp;
      temp = work.KKT[30]-work.L[0]*work.d[0]*work.L[0]-1*work.d[15]*1;
      residual += temp*temp;
      temp = work.KKT[32]-work.L[1]*work.d[1]*work.L[1]-1*work.d[16]*1;
      residual += temp*temp;
      temp = work.KKT[34]-work.L[2]*work.d[2]*work.L[2]-1*work.d[17]*1;
      residual += temp*temp;
      temp = work.KKT[36]-work.L[3]*work.d[3]*work.L[3]-1*work.d[18]*1;
      residual += temp*temp;
      temp = work.KKT[38]-work.L[4]*work.d[4]*work.L[4]-1*work.d[19]*1;
      residual += temp*temp;
      temp = work.KKT[40]-work.L[5]*work.d[5]*work.L[5]-1*work.d[20]*1;
      residual += temp*temp;
      temp = work.KKT[42]-work.L[6]*work.d[6]*work.L[6]-1*work.d[21]*1;
      residual += temp*temp;
      temp = work.KKT[44]-work.L[7]*work.d[7]*work.L[7]-1*work.d[22]*1;
      residual += temp*temp;
      temp = work.KKT[46]-work.L[8]*work.d[8]*work.L[8]-1*work.d[23]*1;
      residual += temp*temp;
      temp = work.KKT[48]-work.L[9]*work.d[9]*work.L[9]-1*work.d[24]*1;
      residual += temp*temp;
      temp = work.KKT[50]-work.L[10]*work.d[10]*work.L[10]-1*work.d[25]*1;
      residual += temp*temp;
      temp = work.KKT[52]-work.L[11]*work.d[11]*work.L[11]-1*work.d[26]*1;
      residual += temp*temp;
      temp = work.KKT[54]-work.L[12]*work.d[12]*work.L[12]-1*work.d[27]*1;
      residual += temp*temp;
      temp = work.KKT[56]-work.L[13]*work.d[13]*work.L[13]-1*work.d[28]*1;
      residual += temp*temp;
      temp = work.KKT[58]-work.L[14]*work.d[14]*work.L[14]-1*work.d[29]*1;
      residual += temp*temp;
      temp = work.KKT[31]-1*work.d[15]*work.L[15];
      residual += temp*temp;
      temp = work.KKT[33]-1*work.d[16]*work.L[16];
      residual += temp*temp;
      temp = work.KKT[35]-1*work.d[17]*work.L[17];
      residual += temp*temp;
      temp = work.KKT[37]-1*work.d[18]*work.L[18];
      residual += temp*temp;
      temp = work.KKT[39]-1*work.d[19]*work.L[19];
      residual += temp*temp;
      temp = work.KKT[41]-1*work.d[20]*work.L[20];
      residual += temp*temp;
      temp = work.KKT[43]-1*work.d[21]*work.L[21];
      residual += temp*temp;
      temp = work.KKT[45]-1*work.d[22]*work.L[22];
      residual += temp*temp;
      temp = work.KKT[47]-1*work.d[23]*work.L[23];
      residual += temp*temp;
      temp = work.KKT[49]-1*work.d[24]*work.L[24];
      residual += temp*temp;
      temp = work.KKT[51]-1*work.d[25]*work.L[25];
      residual += temp*temp;
      temp = work.KKT[53]-1*work.d[26]*work.L[26];
      residual += temp*temp;
      temp = work.KKT[55]-1*work.d[27]*work.L[27];
      residual += temp*temp;
      temp = work.KKT[57]-1*work.d[28]*work.L[28];
      residual += temp*temp;
      temp = work.KKT[59]-1*work.d[29]*work.L[29];
      residual += temp*temp;
      temp = work.KKT[61]-work.L[30]*work.d[30]*1;
      residual += temp*temp;
      temp = work.KKT[63]-work.L[31]*work.d[31]*1;
      residual += temp*temp;
      temp = work.KKT[65]-work.L[32]*work.d[32]*1;
      residual += temp*temp;
      temp = work.KKT[67]-work.L[33]*work.d[33]*1;
      residual += temp*temp;
      temp = work.KKT[69]-work.L[34]*work.d[34]*1;
      residual += temp*temp;
      temp = work.KKT[71]-work.L[35]*work.d[35]*1;
      residual += temp*temp;
      temp = work.KKT[73]-work.L[36]*work.d[36]*1;
      residual += temp*temp;
      temp = work.KKT[75]-work.L[37]*work.d[37]*1;
      residual += temp*temp;
      temp = work.KKT[77]-work.L[38]*work.d[38]*1;
      residual += temp*temp;
      temp = work.KKT[79]-work.L[39]*work.d[39]*1;
      residual += temp*temp;
      temp = work.KKT[81]-work.L[40]*work.d[40]*1;
      residual += temp*temp;
      temp = work.KKT[83]-work.L[41]*work.d[41]*1;
      residual += temp*temp;
      temp = work.KKT[85]-work.L[42]*work.d[42]*1;
      residual += temp*temp;
      temp = work.KKT[87]-work.L[43]*work.d[43]*1;
      residual += temp*temp;
      temp = work.KKT[89]-work.L[44]*work.d[44]*1;
      residual += temp*temp;
      return residual;
    }
    __device__ void matrix_multiply(double *result, double *source) {
      /* Finds result = A*source. */
      result[0] = work.KKT[60]*source[0]+work.KKT[31]*source[30]+work.KKT[61]*source[45];
      result[1] = work.KKT[62]*source[1]+work.KKT[33]*source[31]+work.KKT[63]*source[45];
      result[2] = work.KKT[64]*source[2]+work.KKT[35]*source[32]+work.KKT[65]*source[45];
      result[3] = work.KKT[66]*source[3]+work.KKT[37]*source[33]+work.KKT[67]*source[45];
      result[4] = work.KKT[68]*source[4]+work.KKT[39]*source[34]+work.KKT[69]*source[45];
      result[5] = work.KKT[70]*source[5]+work.KKT[41]*source[35]+work.KKT[71]*source[45];
      result[6] = work.KKT[72]*source[6]+work.KKT[43]*source[36]+work.KKT[73]*source[45];
      result[7] = work.KKT[74]*source[7]+work.KKT[45]*source[37]+work.KKT[75]*source[45];
      result[8] = work.KKT[76]*source[8]+work.KKT[47]*source[38]+work.KKT[77]*source[45];
      result[9] = work.KKT[78]*source[9]+work.KKT[49]*source[39]+work.KKT[79]*source[45];
      result[10] = work.KKT[80]*source[10]+work.KKT[51]*source[40]+work.KKT[81]*source[45];
      result[11] = work.KKT[82]*source[11]+work.KKT[53]*source[41]+work.KKT[83]*source[45];
      result[12] = work.KKT[84]*source[12]+work.KKT[55]*source[42]+work.KKT[85]*source[45];
      result[13] = work.KKT[86]*source[13]+work.KKT[57]*source[43]+work.KKT[87]*source[45];
      result[14] = work.KKT[88]*source[14]+work.KKT[59]*source[44]+work.KKT[89]*source[45];
      result[15] = work.KKT[0]*source[15]+work.KKT[1]*source[30];
      result[16] = work.KKT[2]*source[16]+work.KKT[3]*source[31];
      result[17] = work.KKT[4]*source[17]+work.KKT[5]*source[32];
      result[18] = work.KKT[6]*source[18]+work.KKT[7]*source[33];
      result[19] = work.KKT[8]*source[19]+work.KKT[9]*source[34];
      result[20] = work.KKT[10]*source[20]+work.KKT[11]*source[35];
      result[21] = work.KKT[12]*source[21]+work.KKT[13]*source[36];
      result[22] = work.KKT[14]*source[22]+work.KKT[15]*source[37];
      result[23] = work.KKT[16]*source[23]+work.KKT[17]*source[38];
      result[24] = work.KKT[18]*source[24]+work.KKT[19]*source[39];
      result[25] = work.KKT[20]*source[25]+work.KKT[21]*source[40];
      result[26] = work.KKT[22]*source[26]+work.KKT[23]*source[41];
      result[27] = work.KKT[24]*source[27]+work.KKT[25]*source[42];
      result[28] = work.KKT[26]*source[28]+work.KKT[27]*source[43];
      result[29] = work.KKT[28]*source[29]+work.KKT[29]*source[44];
      result[30] = work.KKT[1]*source[15]+work.KKT[30]*source[30]+work.KKT[31]*source[0];
      result[31] = work.KKT[3]*source[16]+work.KKT[32]*source[31]+work.KKT[33]*source[1];
      result[32] = work.KKT[5]*source[17]+work.KKT[34]*source[32]+work.KKT[35]*source[2];
      result[33] = work.KKT[7]*source[18]+work.KKT[36]*source[33]+work.KKT[37]*source[3];
      result[34] = work.KKT[9]*source[19]+work.KKT[38]*source[34]+work.KKT[39]*source[4];
      result[35] = work.KKT[11]*source[20]+work.KKT[40]*source[35]+work.KKT[41]*source[5];
      result[36] = work.KKT[13]*source[21]+work.KKT[42]*source[36]+work.KKT[43]*source[6];
      result[37] = work.KKT[15]*source[22]+work.KKT[44]*source[37]+work.KKT[45]*source[7];
      result[38] = work.KKT[17]*source[23]+work.KKT[46]*source[38]+work.KKT[47]*source[8];
      result[39] = work.KKT[19]*source[24]+work.KKT[48]*source[39]+work.KKT[49]*source[9];
      result[40] = work.KKT[21]*source[25]+work.KKT[50]*source[40]+work.KKT[51]*source[10];
      result[41] = work.KKT[23]*source[26]+work.KKT[52]*source[41]+work.KKT[53]*source[11];
      result[42] = work.KKT[25]*source[27]+work.KKT[54]*source[42]+work.KKT[55]*source[12];
      result[43] = work.KKT[27]*source[28]+work.KKT[56]*source[43]+work.KKT[57]*source[13];
      result[44] = work.KKT[29]*source[29]+work.KKT[58]*source[44]+work.KKT[59]*source[14];
      result[45] = work.KKT[61]*source[0]+work.KKT[63]*source[1]+work.KKT[65]*source[2]+work.KKT[67]*source[3]+work.KKT[69]*source[4]+work.KKT[71]*source[5]+work.KKT[73]*source[6]+work.KKT[75]*source[7]+work.KKT[77]*source[8]+work.KKT[79]*source[9]+work.KKT[81]*source[10]+work.KKT[83]*source[11]+work.KKT[85]*source[12]+work.KKT[87]*source[13]+work.KKT[89]*source[14];
    }
    __device__ double check_residual(double *target, double *multiplicand) {
      /* Returns the squared 2-norm of lhs - A*rhs. */
      /* Reuses v to find the residual. */
      int i;
      double residual;
      residual = 0;
      matrix_multiply(work.v, multiplicand);
      for (i = 0; i < 15; i++) {
        residual += (target[i] - work.v[i])*(target[i] - work.v[i]);
      }
      return residual;
    }
    __device__ void fill_KKT(void) {
      work.KKT[60] = 2;
      work.KKT[62] = 2;
      work.KKT[64] = 2;
      work.KKT[66] = 2;
      work.KKT[68] = 2;
      work.KKT[70] = 2;
      work.KKT[72] = 2;
      work.KKT[74] = 2;
      work.KKT[76] = 2;
      work.KKT[78] = 2;
      work.KKT[80] = 2;
      work.KKT[82] = 2;
      work.KKT[84] = 2;
      work.KKT[86] = 2;
      work.KKT[88] = 2;
      work.KKT[0] = work.s_inv_z[0];
      work.KKT[2] = work.s_inv_z[1];
      work.KKT[4] = work.s_inv_z[2];
      work.KKT[6] = work.s_inv_z[3];
      work.KKT[8] = work.s_inv_z[4];
      work.KKT[10] = work.s_inv_z[5];
      work.KKT[12] = work.s_inv_z[6];
      work.KKT[14] = work.s_inv_z[7];
      work.KKT[16] = work.s_inv_z[8];
      work.KKT[18] = work.s_inv_z[9];
      work.KKT[20] = work.s_inv_z[10];
      work.KKT[22] = work.s_inv_z[11];
      work.KKT[24] = work.s_inv_z[12];
      work.KKT[26] = work.s_inv_z[13];
      work.KKT[28] = work.s_inv_z[14];
      work.KKT[1] = 1;
      work.KKT[3] = 1;
      work.KKT[5] = 1;
      work.KKT[7] = 1;
      work.KKT[9] = 1;
      work.KKT[11] = 1;
      work.KKT[13] = 1;
      work.KKT[15] = 1;
      work.KKT[17] = 1;
      work.KKT[19] = 1;
      work.KKT[21] = 1;
      work.KKT[23] = 1;
      work.KKT[25] = 1;
      work.KKT[27] = 1;
      work.KKT[29] = 1;
      work.KKT[30] = work.block_33[0];
      work.KKT[32] = work.block_33[0];
      work.KKT[34] = work.block_33[0];
      work.KKT[36] = work.block_33[0];
      work.KKT[38] = work.block_33[0];
      work.KKT[40] = work.block_33[0];
      work.KKT[42] = work.block_33[0];
      work.KKT[44] = work.block_33[0];
      work.KKT[46] = work.block_33[0];
      work.KKT[48] = work.block_33[0];
      work.KKT[50] = work.block_33[0];
      work.KKT[52] = work.block_33[0];
      work.KKT[54] = work.block_33[0];
      work.KKT[56] = work.block_33[0];
      work.KKT[58] = work.block_33[0];
      work.KKT[31] = -1;
      work.KKT[33] = -1;
      work.KKT[35] = -1;
      work.KKT[37] = -1;
      work.KKT[39] = -1;
      work.KKT[41] = -1;
      work.KKT[43] = -1;
      work.KKT[45] = -1;
      work.KKT[47] = -1;
      work.KKT[49] = -1;
      work.KKT[51] = -1;
      work.KKT[53] = -1;
      work.KKT[55] = -1;
      work.KKT[57] = -1;
      work.KKT[59] = -1;
      work.KKT[61] = 1;
      work.KKT[63] = 1;
      work.KKT[65] = 1;
      work.KKT[67] = 1;
      work.KKT[69] = 1;
      work.KKT[71] = 1;
      work.KKT[73] = 1;
      work.KKT[75] = 1;
      work.KKT[77] = 1;
      work.KKT[79] = 1;
      work.KKT[81] = 1;
      work.KKT[83] = 1;
      work.KKT[85] = 1;
      work.KKT[87] = 1;
      work.KKT[89] = 1;
    }


};



__device__ int getGlobalIdx_1D_1D(){
    return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ int getGlobalIdx_2D_1D(){
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;

    return threadId;
}


__global__ void load_scalar_buffer(double* ScalarBuffer, double* S, double* U, int numRows, int numCentroids){
      int k = blockIdx.x;
      int tid = threadIdx.x;



      for(int i = tid; i < numRows; i += blockDim.x){
          ScalarBuffer[deref(k, i, numRows)] = S[deref(i, k, numCentroids)] * U[deref(i, k, numCentroids)];
      }

      __syncthreads();

}


__global__ void calculate_centroids(double* DataMatrix, double* V, double* ScalarBuffer, int numRows, int numFeatures){
    extern __shared__ double localbuffer[];

    int k = blockIdx.x;
    int f = blockIdx.y;
    int tid = threadIdx.x;



    for(int i = tid; i < numRows; i += blockDim.x){
          localbuffer[i] = ScalarBuffer[deref(k, i, numRows)] * DataMatrix[deref(i,f,numFeatures)];
    }

    __syncthreads();



    for(int step = 1; step < numRows; step*=2){

        while( (tid+step < numRows) ){
            int ndx = 2 * step * tid;
            if( ndx + step < numRows){
                localbuffer[ndx] += localbuffer[ndx+step];
            }
            tid += blockDim.x;
        }
        tid = threadIdx.x;
        __syncthreads();
    }

    __syncthreads();



    V[deref(k,f, numRows)] = localbuffer[0];


    // if( k == 0 && f < numFeatures && tid == 0){
    //     printf("V[%d][%d] = %f\n", k, f, V[deref(k, f, numRows)]);
    // }



}

__global__ void find_centroids(double* DataMatrix, double* V, double* ScalarBuffer, int numRows, int numFeatures){
    int gtid = getGlobalIdx_2D_1D();

    int k = blockIdx.x;
    int i = threadIdx.x;


    // if( k == 0 && i < numFeatures){
    //     printf("V[%d][%d] = %f\n", k, i, V[deref(k, i, numRows)]);
    // }


    //Sum working
    for(int step = 1; step < numRows; step*=2){

        while( (i+step < numRows) ){

            int ndx = 2 * step * i;
            if(ndx+step < numRows){
              ScalarBuffer[deref(k, ndx, numRows)] += ScalarBuffer[deref(k, ndx+step, numRows)];
            }
            i += blockDim.x;
        }

        i = threadIdx.x;
        __syncthreads();
    }
    __syncthreads();



    //
    // if( k < numCentroids && i == 0){
    //     printf("ScalarBuffer[%d][%d] = %f\n", k, i, ScalarBuffer[deref(k, i, numRows)]);
    // }

    if(ScalarBuffer[deref(k, 0, numRows)] != 0.0){
        if(i < numFeatures){
            V[deref(k, i, numFeatures)] = V[deref(k, i, numFeatures)] * (1/ScalarBuffer[deref(k, 0, numRows)]);
        }
    }

    __syncthreads();

    // if( k == 0 && i < numFeatures){
    //     printf("V[%d][%d] = %f\n", k, i, V[deref(k, i, numRows)]);
    // }



}


__global__ void init_S(double* S, int numCols){
    int i = blockIdx.x;
    int k = threadIdx.x;

    S[deref(i,k,numCols)] = 1.0;
}

__global__ void build_h_matrix(double* H, double* DataMatrix, double* S, double* Centroids){
      int gtid = getGlobalIdx_2D_1D();

      int k = blockIdx.x;
      int i = blockIdx.y;
      int f = threadIdx.x;

      int numCentroids = gridDim.x;
      int numFeatures = blockDim.x;

      __shared__ double buffer[50]; //unlikely to have more than 50 features

      //Get Square Subtracted vectors
      buffer[f] = (DataMatrix[deref(i,f,numFeatures)] - Centroids[deref(k,f,numFeatures)]) * (DataMatrix[deref(i,f,numFeatures)] - Centroids[deref(k,f,numFeatures)]) ;

      __syncthreads();

      //Do a hekin good sum
      for(int step = 1; step <= numFeatures; step = step*2){
          while(f + step <= numFeatures){
              buffer[f] += buffer[f+step];
              f += blockDim.x;
          }
          f = threadIdx.x;
          __syncthreads();
      }

      if(f == 0){
          H[deref(i,k,numCentroids)] = buffer[f] * S[deref(i,k,numCentroids)];
      }


}

__global__ void update_membership_matrix( double* U, double* U_GPU, double* H, double RegParam, int numClusters, int numRows){
    /* Get Specific Thread Assignment Data */
    int tid = getGlobalIdx_1D_1D();
    int tidy = threadIdx.y;



    if(tid < numRows){
        solver_scope solver(tid);

        solver.set_defaults();  // Set basic algorithm parameters.
        solver.setup_indexing();


        //load one line of the h matrix
        //calculate the h_tilde in here
        double multiplicand = (-1/(2*RegParam));
        for(int i = tidy; i < numClusters; i += blockDim.y){
            solver.params.Hi[i] = H[deref(tid,i, numClusters)] * multiplicand;
        }


        // Solve our problem at high speed!
        solver.solve();


        //use_solution(vars, params, tid);
        for(int i = tidy; i < numClusters; i += blockDim.y){
            U[deref(tid, i, numClusters)] = (double) solver.vars.Ui[i];
            U_GPU[deref(tid, i, numClusters)] = (double) solver.vars.Ui[i];
        }

    }
}
